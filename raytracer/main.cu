#include "hip/hip_runtime.h"
#include <common/common.h>

#include <iostream>

__global__ void testKernel() {
    printf("test kernel");
}

int main() {
    int clusterSize{};
    hipLaunchConfig_t launchConfig{};
    
    size_t dynamicSmemSize{};
    hipError_t err1 = cudaOccupancyAvailableDynamicSMemPerBlock(&dynamicSmemSize, testKernel, 1, 1024);

    int numBlocks{};
	hipError_t err2 = hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, &testKernel, 4, dynamicSmemSize);

    printf("dynamic shared mem: %zu\n", dynamicSmemSize);
    printf("num blocks: %d\n", numBlocks);
    return 0;
}
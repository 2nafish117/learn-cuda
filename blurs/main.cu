#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <chrono>
#include <random>
#include <cassert>

#include <clog/log.h>
#include <stb/stb_image.h>

class ScopedTimer final {
public:
	ScopedTimer(std::string_view name) 
		: m_name(name)
	{
		using namespace std::chrono;
		using namespace std::chrono_literals;

		m_start = high_resolution_clock::now();
	}

	~ScopedTimer() {
		using namespace std::chrono;
		using namespace std::chrono_literals;

		m_stop = high_resolution_clock::now();
		log_info("%s elapsed %f\n", m_name.data(), Elapsed());
	}

private:
	inline double Elapsed() {
		using namespace std::chrono;
		using namespace std::chrono_literals;

		duration<double> duration = m_stop - m_start;
		return duration.count();
	}

	std::string_view m_name;
	std::chrono::high_resolution_clock::time_point m_start{};
	std::chrono::high_resolution_clock::time_point m_stop{};
};

#define SCOPED_TIMER(name) ScopedTimer hidden_scopedTimer(name)

void cudaErrorPrint(hipError_t err) {
	if(err != hipSuccess) {
		const char* errStr = hipGetErrorString(err);
		const char* errName = hipGetErrorName(err);
		log_error("[cuda error %d] %s %s\n", err, errName, errStr);
	}
}

#define CUDA_CHECK(expr) {			\
	hipError_t err = (expr);		\
	cudaErrorPrint(err);			\
}

int main() {
	return 0;
}